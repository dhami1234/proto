#include "hip/hip_runtime.h"
/*
  Copyright Marcin Krotkiewski, University of Oslo, 2012
*/

#define stencil_3x3(c0, c1, c2, shm, tx, ty, bx)			\
  c0*((shm)[tx+0+(ty+0)*bx]) +						\
  c1*((shm)[tx-1+(ty+0)*bx]  + (shm)[tx+1+(ty+0)*bx] + (shm)[tx+0+(ty-1)*bx] + (shm)[tx+0+(ty+1)*bx]) + \
  c2*((shm)[tx-1+(ty-1)*bx]  + (shm)[tx+1+(ty-1)*bx] + (shm)[tx-1+(ty+1)*bx] + (shm)[tx+1+(ty+1)*bx])
  

__device__ inline mfloat stencil_3x3_function(mfloat c0, mfloat c1, mfloat c2, mfloat* shm,
                                uint tx, uint ty, uint bx)
{
  mfloat rtn = 0;
  rtn+=  c0*((shm)[tx+0+(ty+0)*bx]);
  rtn+=  c1*((shm)[tx-1+(ty+0)*bx]  + (shm)[tx+1+(ty+0)*bx] + (shm)[tx+0+(ty-1)*bx] + (shm)[tx+0+(ty+1)*bx]);
  rtn += c2*((shm)[tx-1+(ty-1)*bx]  + (shm)[tx+1+(ty-1)*bx] + (shm)[tx-1+(ty+1)*bx] + (shm)[tx+1+(ty+1)*bx]);
  return rtn;
}
  
  
#define stencil_3x3_reg(c0, c1, c2)					\
  c0*r5 +								\
  c1*(r2+r4+r6+r8) +							\
  c2*(r1+r3+r7+r9)

__device__ inline void push_regs_3x3(mfloat *shm, mfloat *r, uint tx, uint ty, uint bx)
{
    r[0] = shm[tx+ty*bx];
    if (tx<16) // ensures all reads are from same bank
        r[3] = shm[tx+16+(ty+1)*bx];
    else
        r[3] = shm[tx-16+(ty+1)*bx];
    r[6] = shm[tx+(ty+2)*bx];
}

__device__ inline mfloat calc_regs_3x3(mfloat *r, mfloat c0, mfloat c1, mfloat c2) {
    return c0*r[4] + c1*(r[1]+r[3]+r[5]+r[7]) + c2*(r[0]+r[2]+r[6]+r[8]);
}

#define push_regs_exp(shm, bx)			\
  {						\
    r1=(shm)[tx-1+(ty-1)*bx];			\
    r2=(shm)[tx+0+(ty-1)*bx];			\
    r3=(shm)[tx+1+(ty-1)*bx];			\
						\
    r4=(shm)[tx-1+(ty+0)*bx];			\
    r5=(shm)[tx+0+(ty+0)*bx];			\
    r6=(shm)[tx+1+(ty+0)*bx];			\
	    					\
    r7=(shm)[tx-1+(ty+1)*bx];			\
    r8=(shm)[tx+0+(ty+1)*bx];			\
    r9=(shm)[tx+1+(ty+1)*bx];			\
  }						\

namespace cg = cooperative_groups;

__global__ void stencil27_symm_exp(mfloat *in, mfloat *out, uint dimx, uint dimy, uint dimz, uint kstart, uint kend)
{
  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const  int ix = blockIdx.x*blockDim.x + threadIdx.x;	
  const  int iy = blockIdx.y*blockDim.y + threadIdx.y;

  const uint ti = threadIdx.y*blockDim.x + threadIdx.x;
  const uint pad = 32/sizeof(mfloat); // halos to left & right of interior require 32 byte memory transaction
  const uint bx= blockDim.x+2*pad;
  const uint txe= ti%bx; // this thread's block-relative x-axis index for first read
  const uint tye= ti/bx; // this thread's block-relative y-axis index for first read
  const uint txe2= (ti+blockDim.x*blockDim.y)%bx; // because of halos, each thread reads two values
  const uint tye2= (ti+blockDim.x*blockDim.y)/bx;
  int  ixe= blockIdx.x*blockDim.x + txe - pad; // this thread's global x-axis index for first read
  int  iye= blockIdx.y*blockDim.y + tye - 1;
  int  ixe2= blockIdx.x*blockDim.x + txe2 - pad;
  int  iye2= blockIdx.y*blockDim.y + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(ixe2<0)      ixe2 += dimx;
  if(ixe2>dimx-1) ixe2 -= dimx;

  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];
  uint i1, i2;

  cg::thread_block block = cg::this_thread_block();
  extern __shared__ mfloat shm[];			

  i1 = ixe+iye*dimx;
  i2 = ixe2+iye2*dimy;

  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  block.sync();
   t1 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, bx);
  block.sync();

  i1 += dimx*dimy;
  i2 += dimx*dimy;

  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  block.sync();
  t2 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, bx);
  t1+= stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, bx);
  block.sync();

  for(uint kk=kstart; kk<kend; kk++){

    block.sync();

    i1 += dimx*dimy;
    i2 += dimx*dimy;

    shm[txe +tye *bx] = in[i1];
    shm[txe2+tye2*bx] = in[i2];

    block.sync();
    t3 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, bx);

    out[ix + iy*dimx + kk*dimx*dimy] = t1 + t3;
    t1 = t2 + stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, bx);
    t2 = t3;
  }
}


__global__ void stencil27_symm_exp_prefetch(mfloat *in, mfloat *out, uint dimx, uint dimy, uint dimz, uint kstart, uint kend)
{
  mfloat r[9];

  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const  int ix = blockIdx.x*blockDim.x + threadIdx.x; // global x index
  const  int iy = blockIdx.y*blockDim.y + threadIdx.y; // global y index

  const uint ti = threadIdx.y*blockDim.x + threadIdx.x; // local 2D index
  const uint pad = 32/sizeof(mfloat);
  const uint bx= blockDim.x+2*pad;
  const uint txe= (2*ti)%bx; // local x-coordinate of float2 read in padded space
  const uint tye= (2*ti)/bx; // local y-coordinate of float2 read in padded space
  const uint wi = ty*2*32 + tx; // 2D index of first shmem write
  const uint wxe= wi%bx; // x-coordinate of 1st shmem write
  const uint wye= wi/bx; // y-coordinate of 1st shmem write
  const uint wxe2= (wi+32)%bx; // x-coordinate of 2nd shmem write
  const uint wye2= (wi+32)/bx; // y-coordinate of 2nd shmem write
//  const uint txe2= (ti+blockDim.x*blockDim.y)%bx;
//  const uint tye2= (ti+blockDim.x*blockDim.y)/bx;
  int  ixe= blockIdx.x*blockDim.x + txe - pad; // global x-coordinate of float2 read in unpadded space
  int  iye= blockIdx.y*blockDim.y + tye - 1; // global y-coordinate of float2 read in unpadded space
//  int  ixe2= blockIdx.x*blockDim.x + txe2 - pad;
//  int  iye2= blockIdx.y*blockDim.y + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
//  if(ixe2<0)      ixe2 += dimx;
//  if(ixe2>dimx-1) ixe2 -= dimx;

  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
//  if(iye2<0)      iye2 += dimy;
//  if(iye2>dimy-1) iye2 -= dimy;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];

  uint i1; //, i2;
//  mfloat s[2];
  float2 read;

  cg::thread_block block = cg::this_thread_block();						
  cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
  extern __shared__ mfloat sh1[];
  mfloat *sh2 = sh1 + 2*blockDim.x*blockDim.y; // offset by one slice of padded data (same as bx*8)
  mfloat *shm[2] = { sh1, sh2 };

  i1 = ixe+iye*dimx;
//  i2 = ixe2+iye2*dimx;
  read = reinterpret_cast<float2*>(in)[i1/2]; // reads 8 contiguous bytes of data
//  shm[0][txe +tye *bx] = in[i1]; // load and store first slice
//  shm[0][txe2+tye2*bx] = in[i2];
  if (warp.thread_rank() < 16) // get first value from higher-ranked thread
    read.y = warp.shfl_xor(read.x, 16);
  else // get second value from lower-ranked thread
    read.x = warp.shfl_xor(read.y, 16);
  shm[0][wxe +wye *bx] = read.x; // read's two values should be in same lane
  shm[0][wxe2+wye2*bx] = read.y;

  i1 += dimx*dimy;
//  i2 += dimx*dimy;
  read = reinterpret_cast<float2*>(in)[i1/2]; // load second slice
//  s[0] = in[i1];
//  s[1] = in[i2];
  block.sync();
  push_regs_3x3(shm[0]+pad, &r[1], tx, ty, bx); // push middle of first slice 
  r[4] = warp.shfl_xor(r[4], 16); // thread gets its correct center value
  if (tx==0)
      push_regs_3x3(shm[0]+pad, &r[0], tx-1, ty, bx); // push block's left slice
  if (tx==31)
      push_regs_3x3(shm[0]+pad, &r[2], tx+1, ty, bx); // push block's right slice
  if (warp.thread_rank() < 16)
    read.y = warp.shfl_xor(read.x, 16); // threads now have data 32 bytes apart
  else
    read.x = warp.shfl_xor(read.y, 16);
  shm[1][wxe +wye *bx] = read.x; // store second slice
  shm[1][wxe2+wye2*bx] = read.y;
  for (int i = 0; i < 3; i++) {
      r[3*i+0] = warp.shfl_up(r[3*i+1], 1); // fill left halo slice
      r[3*i+2] = warp.shfl_down(r[3*i+1], 1); // fill right halo slice
  }
  t1 = calc_regs_3x3(r, C1, C2, C3); // calculate fist slice

  i1 += dimx*dimy;
//  i2 += dimx*dimy;
  read = reinterpret_cast<float2*>(in)[i1/2];
//  s[0] = in[i1];
//  s[1] = in[i2];
  block.sync();
  push_regs_3x3(shm[1]+pad, &r[1], tx, ty, bx);
  r[4] = warp.shfl_xor(r[4], 16);
  if (tx==0)
      push_regs_3x3(shm[1]+pad, &r[0], tx-1, ty, bx);
  if (tx==31)
      push_regs_3x3(shm[1]+pad, &r[2], tx+1, ty, bx);
  if (warp.thread_rank() < 16)
    read.y = warp.shfl_xor(read.x, 16); // threads now have data 32 bytes apart
  else
    read.x = warp.shfl_xor(read.y, 16);
  shm[0][wxe +wye *bx] = read.x; // store second slice
  shm[0][wxe2+wye2*bx] = read.y;
  for (int i = 0; i < 3; i++) {
      r[3*i+0] = warp.shfl_up(r[3*i+1], 1); // fill left halo slice
      r[3*i+2] = warp.shfl_down(r[3*i+1], 1); // fill right halo slice
  }
  t2 = calc_regs_3x3(r, C1, C2, C3);
  t1+= calc_regs_3x3(r, C0, C1, C2);

  uint j=0, k=1, kk;
  for(kk=kstart; kk<kend-1; kk++){

    i1 += dimx*dimy;
//    i2 += dimx*dimy;
    read = reinterpret_cast<float2*>(in)[i1/2];
//    s[0] = in[i1];
//    s[1] = in[i2];

    block.sync();
    push_regs_3x3(shm[j]+pad, &r[1], tx, ty, bx);
    r[4] = warp.shfl_xor(r[4], 16);
    if (tx==0)
      push_regs_3x3(shm[j]+pad, &r[0], tx-1, ty, bx);
    if (tx==31)
      push_regs_3x3(shm[j]+pad, &r[2], tx+1, ty, bx);
   
    if (warp.thread_rank() < 16)
      read.y = warp.shfl_xor(read.x, 16); // threads now have data 32 bytes apart
    else
      read.x = warp.shfl_xor(read.y, 16);

    shm[k][wxe +wye *bx] = read.x; // store second slice
    shm[k][wxe2+wye2*bx] = read.y;

    for (int i = 0; i < 3; i++) {
      r[3*i+0] = warp.shfl_up(r[3*i+1], 1); // fill left halo slice
      r[3*i+2] = warp.shfl_down(r[3*i+1], 1); // fill right halo slice
    }

    t3 = calc_regs_3x3(r, C1, C2, C3);
    out[ix + iy*dimx + kk*dimx*dimy] = t1 + t3;
    t1 = t2 + calc_regs_3x3(r, C0, C1, C2);
    t2 = t3;

    j=1-j; k=1-k;
  }

  block.sync();  
  push_regs_3x3(shm[j]+pad, &r[1], tx, ty, bx);
  r[4] = warp.shfl_xor(r[4], 16);
  if (tx==0)
    push_regs_3x3(shm[j]+pad, &r[0], tx-1, ty, bx);
  if (tx==31)
    push_regs_3x3(shm[j]+pad, &r[2], tx+1, ty, bx);
   
  for (int i = 0; i < 3; i++) {
    r[3*i+0] = warp.shfl_up(r[3*i+1], 1); // fill left halo slice
    r[3*i+1] = warp.shfl_down(r[3*i+1], 1); // fill right halo slice
  }

  t3 = calc_regs_3x3(r, C1, C2, C3);
  out[ix + iy*dimx + kk*dimx*dimy] = t1 + t3;
}


__global__ void stencil27_symm_exp_new(mfloat *in, mfloat *out, uint dimx, uint dimy, uint dimz, uint kstart, uint kend)
{
  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const uint ix = blockIdx.x*32 + threadIdx.x; // 32 = blockDim.x
  const uint iy = blockIdx.y*6 + threadIdx.y; // 6 = blockDim.y
  const uint ti = threadIdx.y*32 + threadIdx.x;
  const uint pad = 32/sizeof(mfloat);
  const uint width = 32+2*pad; // width of slice, including halos
  const uint tye= ti/width;
  const uint txe= ti-tye*width;
  const uint tye2=tye+4; // including halos, slice has 8 rows, so tye2 is 4 rows below tye

  int  ixe = blockIdx.x*32 + txe  - pad;
  int  iye = blockIdx.y*6  + tye  - 1;
  int  iye2= blockIdx.y*6  + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  uint i1, i2;
  
  i1 = ixe+iye*dimx ;
  i2 = ixe+iye2*dimx ;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];

  cg::thread_block block = cg::this_thread_block();
  extern __shared__ mfloat shm[];			

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  block.sync();
  t1 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, width);
  block.sync();

  i1 += dimx*dimy;
  i2 += dimx*dimy;

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  block.sync();
  t2 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, width);
  t1+= stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, width);
  block.sync();

  for(uint kk=kstart; kk<kend; kk++){

    block.sync();

    i1 += dimx*dimy;
    i2 += dimx*dimy;

    shm[txe +tye *width] = in[i1];
    shm[txe+tye2*width] = in[i2];

    block.sync();
    t3 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, width);

    out[ix + iy*dimx + kk*dimx*dimy] = t1 + t3;
    t1 = t2 + stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, width);
    t2 = t3;
  }
}


__global__ void stencil27_symm_exp_prefetch_new(mfloat *in, mfloat *out, uint dimx, uint dimy, uint dimz, uint kstart, uint kend)
{
  mfloat r1, r2, r3, r4, r5, r6, r7, r8, r9;

  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const uint ix = blockIdx.x*32 + threadIdx.x;
  const uint iy = blockIdx.y*6  + threadIdx.y;
  const uint ti = threadIdx.y*32 + threadIdx.x;
  const uint pad= 32/sizeof(mfloat);
  const uint width = 32+2*pad;
  const uint tye= ti/width;
  const uint txe= ti-tye*width;
  const uint tye2=tye+4;

  int  ixe = blockIdx.x*32 + txe  - pad;
  int  iye = blockIdx.y*6  + tye  - 1;
  int  iye2= blockIdx.y*6  + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  uint i1, i2;
  
  i1 = ixe+iye*dimx ;
  i2 = ixe+iye2*dimx ;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];

  cg::thread_block block = cg::this_thread_block();
  extern __shared__ mfloat shm[];

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  block.sync();  
  push_regs_exp(shm+pad+width, width);  
  block.sync();

  i1 += dimx*dimy;
  i2 += dimx*dimy;

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  t1 = stencil_3x3_reg(C1, C2, C3);

  block.sync();  
  push_regs_exp(shm+pad+width, width);  
  block.sync();

  i1 += dimx*dimy;
  i2 += dimx*dimy;

  shm[txe +tye*width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  t2 = stencil_3x3_reg(C1, C2, C3);
  t1+= stencil_3x3_reg(C0, C1, C2);

  for(uint kk=kstart; kk<kend-1; kk++){

    block.sync();  
    push_regs_exp(shm+pad+width, width);  
    block.sync();

    i1 += dimx*dimy;
    i2 += dimx*dimy;

    shm[txe +tye *width] = in[i1];
    shm[txe+tye2*width] = in[i2];

    t3 = stencil_3x3_reg(C1, C2, C3);

    out[ix + iy*dimx + kk*dimx*dimy] = t1 + t3;
    t1 = t2 + stencil_3x3_reg(C0, C1, C2);
    t2 = t3;
  }

  block.sync();  
  push_regs_exp(shm+pad+width, width);  
  block.sync();

  out[ix + iy*dimx + (kend-1)*dimx*dimy] = t1 + stencil_3x3_reg(C1, C2, C3);
}
